#include "hip/hip_runtime.h"
#include "include/nbody.cpp"

__device__ void update_positions(particle *d_bodies,float dt,int n){

	int i = threadIdx.x + blockDim.x * blockIdx.x;

	if(i < n){
		d_bodies[i].ax0 = d_bodies[i].ax;
		d_bodies[i].ay0 = d_bodies[i].ay;
		d_bodies[i].az0 = d_bodies[i].az;
	
		d_bodies[i].x +=  dt*d_bodies[i].vx + 0.5*dt*dt*d_bodies[i].ax0; 
		d_bodies[i].y +=  dt*d_bodies[i].vy + 0.5*dt*dt*d_bodies[i].ay0;
		d_bodies[i].z +=  dt*d_bodies[i].vz + 0.5*dt*dt*d_bodies[i].az0;
	}
}

__device__ void update_velocities(particle *d_bodies, float dt,int n){
	
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	if(i < n){
		d_bodies[i].vx +=  0.5*dt*(d_bodies[i].ax+d_bodies[i].ax0); 
		d_bodies[i].vy +=  0.5*dt*(d_bodies[i].ay+d_bodies[i].ay0);
		d_bodies[i].vz +=  0.5*dt*(d_bodies[i].az+d_bodies[i].az0);
		
		d_bodies[i].ax0 = d_bodies[i].ax;
		d_bodies[i].ay0 = d_bodies[i].ay;
		d_bodies[i].az0 = d_bodies[i].az;
	}
}

__device__ void reset_accelerations(particle *d_bodies, int n){

	int i = threadIdx.x + blockDim.x * blockIdx.x;
	
	if(i < n){
		d_bodies[i].ax = 0.0;
		d_bodies[i].ay = 0.0;
		d_bodies[i].az = 0.0;
	}

}

__device__ void update_accelerations(particle *d_bodies,int n){

	int j;
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	float dx=0.0,dy=0.0,dz=0.0;
	float r=0.0, r3=0.0, invr3=0.0,f=0.0;

	if (i < n){	
	    for(j=0; j<n; j++){
	       	if(j != i){
	
				dx=d_bodies[i].x-d_bodies[j].x;
				dy=d_bodies[i].y-d_bodies[j].y;
				dz=d_bodies[i].z-d_bodies[j].z;
	
				r = dx*dx + dy*dy + dz*dz;
				r3 = r*r*r;
				invr3 = 1.0/sqrt(r3);
	
				f=d_bodies[j].m*invr3;
	
				d_bodies[i].ax -=  f*dx;
				d_bodies[i].ay -=  f*dy;
				d_bodies[i].az -=  f*dz;
	
			}
		}
	}

}

__global__ void nbody(particle *d_bodies,float ite, float dt, int n){

	float t;
	reset_accelerations(d_bodies,n);
	__syncthreads();
	update_accelerations(d_bodies, n);
	__syncthreads();

	for (t = 0.0; t < ite; t += dt){

		update_positions(d_bodies,dt,n);
		__syncthreads();

		reset_accelerations(d_bodies,n);
		__syncthreads();

		update_accelerations(d_bodies,n);
		__syncthreads();

		update_velocities(d_bodies,dt,n);
		__syncthreads();
	}
	// end nbody algorithm


}

int main(int argc, const char *argv[])
{
	bool file = true;

	checkAndInit(argc, argv);

	initPopulation(file,argv[2]);
	// time stuff
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	// end time stuff

	particle *d_bodies;
	int size = n*sizeof(particle);
	hipMalloc((void**) &d_bodies, size);
	hipMemcpy(d_bodies,bodies,size,hipMemcpyHostToDevice);

	nbody<<< blockNum, threadsPerBlock >>> (d_bodies,ite,dt,n);

	hipDeviceSynchronize();	

	hipMemcpy(bodies,d_bodies,size,hipMemcpyDeviceToHost);
	hipFree(d_bodies);

	// time stuff
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	cout << time/1000 << endl;
	// end time stuff
	return 0;
}
